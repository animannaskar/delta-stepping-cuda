#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <climits>
#include <fstream>
#include <sstream>
#include <chrono>

using namespace std;

const int max_dist = 1e6;

__global__
void relax(int* d_node_lock, int *d_all_bucket, int *d_bucket_id, int *d_n, int *d_head_bucket, int *d_tail_bucket, int *d_c, int *d_r, int *d_dist, int *d_delta){
    int thread  = blockIdx.x * blockDim.x + threadIdx.x;

    if(thread >= d_tail_bucket[(*d_bucket_id)] - d_head_bucket[(*d_bucket_id)]) return;

    int index = d_head_bucket[(*d_bucket_id)] + thread;
    index %= (*d_n);

//    printf("index %d\n", index);

    int d_u = d_all_bucket[(*d_bucket_id) * (*d_n) + index];

//    printf("%d d_u %d\n", thread, d_u);

    for(int i = d_r[d_u]; i < d_r[d_u+1]; i+=2){
        int d_v = d_c[i];
        int d_v_dist, d_v_bucket;

        bool updated_dist = 0;

        bool leave1 = 0;
        while (!leave1) {
            if(!atomicCAS(d_node_lock + d_v,0,1)){
              d_v_dist = d_dist[d_u]+d_c[i+1];
              d_v_bucket = d_v_dist/(*d_delta);

//              printf("%d d_v %d d_v_dist %d\n", thread, d_v, d_v_dist);

              if(d_dist[d_v] > d_v_dist){
                d_dist[d_v] = d_v_dist;
                updated_dist = 1;
              }
              leave1 = 1;
              d_node_lock[d_v] = 0;

              __threadfence();
            }
        }
        if(updated_dist){
          int tail = atomicInc((unsigned int*)(d_tail_bucket+d_v_bucket), (unsigned int)(*d_n));   //all vars should be int and int only
          d_all_bucket[d_v_bucket * (*d_n) + tail] = d_v;
          __threadfence();

//          bool leave2 = 0;
//          while (!leave2) {
//              if(!atomicCAS(d_lock + d_v_bucket,0,1)){

//                d_all_bucket[d_v_bucket * (*d_n) + d_tail_bucket[d_v_bucket]] = d_v;

//                d_tail_bucket[d_v_bucket]++;
//                d_tail_bucket[d_v_bucket]%=(*d_n);

//                leave2 = 1;                         //why doesn't break work???
//                d_lock[d_v_bucket] = 0;

//                __threadfence();
//              }
//          }
        }
    }
}

void read_adj_list(const string& filename, thrust::host_vector<int>& h_r, thrust::host_vector<int>& h_c) {
    ifstream file(filename);
    string line;

    int row_pointer = 0;
    while (getline(file, line)) {
        h_r.push_back(row_pointer);

        istringstream iss(line);
        int neighbor, weight;

        while (iss >> neighbor >> weight) {
            h_c.push_back(neighbor);
            h_c.push_back(weight);
            row_pointer+=2;
        }
    }
    h_r.push_back(row_pointer);
}

string get_arg(int argc, char *argv[], string arg_name){
    for(int i = 1; i < argc; i++){
        string arg_i = argv[i];
        if(arg_i == arg_name && i+1 < argc){
            string arg = argv[i+1];
            return arg;
        }
    }
    return "";
}

int main(int argc, char *argv[]){
    int src = 0;
    int delta = 3;
    int cnt_bucket = 32;
    int block_size = 1024;
    string graph_name = "100K";

    string arg;
    int temp;

    arg = get_arg(argc,argv,"-cb");
    if(arg != "" && (temp=stoi(arg))!=0) cnt_bucket = temp;

    arg = get_arg(argc,argv,"-del");
    if(arg != "" && (temp=stoi(arg))!=0) delta = temp;

    arg = get_arg(argc,argv,"-bs");
    if(arg != "" && (temp=stoi(arg))!=0) block_size = temp;

    arg = get_arg(argc,argv,"-g");
    if(arg != "") graph_name = arg;

    string filename = "connected_graph_"+graph_name+".txt";
    string output_file_name = "output_"+graph_name+"_"+to_string(block_size)+"_"+to_string(delta)+".txt";

    thrust::host_vector<int> h_r, h_c;

    read_adj_list(filename, h_r, h_c);

    int n = h_r.size()-1;

    // Some dummy vector to wake up device
    thrust::device_vector<int> dummy_vec (1000000, 1);

    auto start_time_cpy = chrono::high_resolution_clock::now();

    int *d_n;
    hipMalloc((void**)&d_n , sizeof(int));
    hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

    int *d_delta;
    hipMalloc((void**)&d_delta, sizeof(int));
    hipMemcpy(d_delta, &delta, sizeof(int), hipMemcpyHostToDevice);

    int *d_b_size;
    hipMalloc((void**)&d_b_size, sizeof(int));

    int *d_curr;
    hipMalloc((void**)&d_curr, sizeof(int));


//    thrust::device_vector<int> d_lock(cnt_bucket,0);
    thrust::device_vector<int> d_node_lock(n,0);


    thrust::device_vector<int> d_c(h_c);
    thrust::device_vector<int> d_r(h_r);

    thrust::device_vector<int> d_dist(n, max_dist);
    d_dist[src] = 0;

    thrust::device_vector<int> d_all_bucket(cnt_bucket*n);
    thrust::device_vector<int> d_head_bucket(cnt_bucket,0);
    thrust::device_vector<int> d_tail_bucket(cnt_bucket,0);   //next ptr after tail

    d_all_bucket[0*n + 0] = src;
    d_tail_bucket[0]++;                                       //use relax for this?

    auto stop_time_cpy = chrono::high_resolution_clock::now();
    auto duration_cpy = chrono::duration_cast<chrono::microseconds>(stop_time_cpy - start_time_cpy);

    auto start_time_kernel = chrono::high_resolution_clock::now();

    for(int curr = 0; curr < cnt_bucket; curr++){             //reusing empty buckets???
//    printf("bucket_id %d\n",curr);
      int b_size;
//      cout << "b_size" << d_tail_bucket[curr] - d_head_bucket[curr] << "\n";
      while(b_size = d_tail_bucket[curr] - d_head_bucket[curr]){

          hipMemcpy(d_b_size, &b_size, sizeof(int), hipMemcpyHostToDevice);
          hipMemcpy(d_curr, &curr, sizeof(int), hipMemcpyHostToDevice);

          relax<<<(b_size/block_size)+1, block_size>>>(thrust::raw_pointer_cast(d_node_lock.data()), thrust::raw_pointer_cast(d_all_bucket.data()), d_curr, d_n, thrust::raw_pointer_cast(d_head_bucket.data()), thrust::raw_pointer_cast(d_tail_bucket.data()), thrust::raw_pointer_cast(d_c.data()), thrust::raw_pointer_cast(d_r.data()), thrust::raw_pointer_cast(d_dist.data()), d_delta);
          hipDeviceSynchronize();

//         printf("bucket \n");
//          for(int i = d_head_bucket[curr]; i < d_tail_bucket[curr]; i++){
//            printf("%d ", (int)d_all_bucket[curr*n + i]);
//          }
//          printf("\n");

          hipError_t err = hipGetLastError();
          if (err) {
              cerr << "Error: " << hipGetErrorString(err) << "\n";
              return 1;
          }

          d_head_bucket[curr] += b_size;
          d_head_bucket[curr] %= n;
      }
    }

    auto stop_time_kernel = chrono::high_resolution_clock::now();
    auto duration_kernel = chrono::duration_cast<chrono::microseconds>(stop_time_kernel - start_time_kernel);

    ofstream output_file(output_file_name);

    if (!output_file.is_open()) {
        cerr << "Error: Unable to open output file." << endl;
        return 1;
    }

    output_file << "Delta Stepping Execution Time Kernel: " << duration_kernel.count()/1000.0 << " milliseconds" << endl;

    output_file << "Delta Stepping Time FULL: " << duration_kernel.count()/1000.0+duration_cpy.count()/1000.0 << " milliseconds" << endl;

    thrust::device_vector<int> dist(d_dist.begin(), d_dist.end());

    for (int i = 0; i < n; ++i) {
        output_file << "Distance to node " << i << ": " << dist[i] << endl;
    }

    output_file.close();

    hipFree(d_n);
    hipFree(d_delta);
    hipFree(d_b_size);
    hipFree(d_curr);

    return 0;
}
